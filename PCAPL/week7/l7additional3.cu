#include <stdio.h>
#include <hip/hip_runtime.h>

//odd-even transposition sort

__global__ void oddSort(int* array, int n)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i % 2 == 1 && i < n - 1) {
        if (array[i] > array[i+1]) {
            int tmp = array[i];
            array[i] = array[i+1];
            array[i+1] = tmp;
        }
    }
}

__global__ void evenSort(int* array, int n)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i % 2 == 0 && i < n - 1) {
        if (array[i] > array[i+1]) {
            int tmp = array[i];
            array[i] = array[i+1];
            array[i+1] = tmp;
        }
    }
}

int main()
{
    const int n = 10;
    int h_array[n] = {7, 2, 9, 4, 6, 1, 3, 8, 5, 0};
    int* d_array;

    hipMalloc(&d_array, n * sizeof(int));
    hipMemcpy(d_array, h_array, n * sizeof(int), hipMemcpyHostToDevice);

    dim3 block(32);
    dim3 grid((n + block.x - 1) / block.x);

    // Odd-even transposition sort loop
    for (int i = 0; i < n; i++) {
        oddSort<<<grid, block>>>(d_array, n);
        evenSort<<<grid, block>>>(d_array, n);
    }

    hipMemcpy(h_array, d_array, n * sizeof(int), hipMemcpyDeviceToHost);

    for (int i = 0; i < n; i++) {
        printf("%d ", h_array[i]);
    }
    printf("\n");

    hipFree(d_array);

    return 0;
}
