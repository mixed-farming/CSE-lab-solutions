#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

__global__ void modify(int *y, int *x, int alpha, int n)
{
    int tid = threadIdx.x;
    if(tid<n)
    {
        y[tid] = alpha * x[tid] + y[tid];
    }
}

int main()
{
    int n=10;
    int y[10]={1,3,9,4,5,2,7,8,6,0},x[10]={0,2,4,6,8,1,3,5,7,9}, *d_y, *d_x;
    int alpha=6;

    hipMalloc(&d_y, n * sizeof(int));
    hipMemcpy(d_y, y, n * sizeof(int), hipMemcpyHostToDevice);
    hipMalloc(&d_x, n * sizeof(int));
    hipMemcpy(d_x, x, n * sizeof(int), hipMemcpyHostToDevice);

    modify<<<1, 256>>>(d_y, d_x, alpha, n);

    int z[10];
    hipMemcpy(z, d_y, n * sizeof(int), hipMemcpyDeviceToHost);

    printf("Modified vector : ");
    for(int i=0;i<n;i++)
    {
        printf("%d ",z[i]);
    }

    hipFree(d_y);
    hipFree(d_x);

    return 0;
}
