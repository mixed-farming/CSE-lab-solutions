#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <hip/hip_runtime.h>
 
__global__ void compute_sine(float* input, float* output, int size) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < size) {
        output[i] = sinf(input[i]);
    }
}
 
int main() {
    int size = 180;//divides the interval 0-2pi into 180 parts each spaced by 1/180
    size_t bytes = size * sizeof(float);
 
    float* input = (float*)malloc(bytes);
    for (int i = 0; i < size; i++) {
        input[i] = (float)i / size * 2 * 3.14159;
    }
 
    float* output = (float*)malloc(bytes);
    float* d_input;
    float* d_output;
    hipMalloc((void**)&d_input, bytes);
    hipMalloc((void**)&d_output, bytes);
    hipMemcpy(d_input, input, bytes, hipMemcpyHostToDevice);
 
    int block_size = 256;
    int num_blocks = (size + block_size - 1) / block_size;
    compute_sine<<<num_blocks, block_size>>>(d_input, d_output, size);
 
    hipMemcpy(output, d_output, bytes, hipMemcpyDeviceToHost);
    for (int i = 0; i < size; i++) {
        printf("sin(%.4f) = %.4f\n", input[i], output[i]);
    }
 
    hipFree(d_input);
    hipFree(d_output);
    free(input);
    free(output);
    return 0;
}
