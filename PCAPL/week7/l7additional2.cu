#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

//selection sort

__global__ void sort(int *a, int *b,int m, int n)
{

 int row = blockIdx.y * blockDim.y + threadIdx.y;
    int start = row * n;
    int end = start + n;

    for (int i = start; i < end - 1; i++) {
        int min_idx = i;
        for (int j = i + 1; j < end; j++) {
            if (a[j] < a[min_idx]) {
                min_idx = j;
            }
        }
        // Swap elements
        int tmp = a[i];
        a[i] = a[min_idx];
        a[min_idx] = tmp;
    }

    // Copy sorted row to output matrix
    for (int i = start; i < end; i++) {
        b[i] = a[i];
    }
}

int main()
{
 int m=3,n=4;
 int a[m][n]={{4,2,3,1},{6,5,4,3},{8,6,7,5}},b[m][n];
 int *d_a,*d_b;
 hipMalloc((void**)&d_a,sizeof(int)*m*n);
 hipMalloc((void**)&d_b,sizeof(int)*m*n);
 hipMemcpy(d_a,a,sizeof(int)*m*n,hipMemcpyHostToDevice);
 int s=2;
 dim3 block(s,s,1);
 dim3 grid((n+s-1)/s,(m+s-1)/s,1);
 sort<<<grid,block>>>(d_a,d_b,m,n);
 hipMemcpy(b,d_b,sizeof(int)*m*n,hipMemcpyDeviceToHost);
 for(int i=0;i<m;i++)
 {
     for(int j=0;j<n;j++)
     {
         printf("%d ",b[i][j]);
     }
  printf("\n");
 }

 hipFree(d_a); hipFree(d_b);
 return 0;
}

