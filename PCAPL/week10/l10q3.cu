#include <stdio.h>
#include <hip/hip_runtime.h>


__global__ void kkernel(int *A, int m, int n)
{
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int sum = 0, digit, power = 1;
    char binnum[10];
    int num, rem, k = 0;
    if ((row > 0 && row < n - 1) && (col > 0 && col < m - 1))
    {
        num = A[row * n + col];
        while (num > 0)
        {
            rem = num % 2;
            if (rem == 1)
                binnum[k++] = '1';
            else
                binnum[k++] = '0';
            num /= 2;
        }
        k--;
        int t;
        printf("before reverse %s\n", binnum);
        for (int i = 0; i <= k / 2; i++)
        {
            t = binnum[i];
            binnum[i] = binnum[k - i];
           binnum[k - i] = t;
        }
        printf("row no %d and col no. %d after reverse %s\n", row, col, binnum);
        for (int i = k; i >= 0; i--)
        {
            if (binnum[i] == '1')
                digit = 0;
            else
                digit = 1;
            sum += digit * power;
            power = power * 10;
        }
        A[row * n + col] = sum;
    }
}
int main()
{
    int A[4][4] = {{1, 2, 3, 4}, {6, 5, 8, 3}, {2, 4, 10, 1}, {9, 1, 2, 5}};
    int i, j;
    int M = 4, N = 4;
    int vsize = (M * N) * sizeof(int);
    int B[vsize];
    int k = 0;
    for (i = 0; i < M; i++)
    {
        for (j = 0; j < N; j++)
           B[k++] = A[i][j];
    }
    int *d_A;
    hipMalloc((void **)&d_A, sizeof(int) * (vsize));
    hipMemcpy(d_A, B, sizeof(int) * (vsize), hipMemcpyHostToDevice);
    float BSIZE = 2;
    dim3 blck(BSIZE, BSIZE, 1);
    dim3 grid(ceil(N / BSIZE), ceil(M / BSIZE), 1);
    kkernel<<<grid, blck>>>(d_A, M, N);
    hipMemcpy(B, d_A, vsize, hipMemcpyDeviceToHost);
    k = 0;
    for (i = 0; i < M; i++)
    {
        for (j = 0; j < N; j++)
            A[i][j] = B[k++];
    }
    printf("\n");
    for (i = 0; i < M; i++)
    {
        for (j = 0; j < N; j++)
            printf("%d ", A[i][j]);
        printf("\n");
    }
    return 0;
}
