#include <stdio.h>
#include <hip/hip_runtime.h>

__global__ void MatModify(char *a, int *b, char *c, int n, int m)
{
    int row = blockIdx.x;
    int col = threadIdx.x;
    int index = row * n + col;
    int startindex=0;
    for(int i=0;i<index;i++)
    {
        startindex+=b[i];
    }
    for(int i=0;i<b[index];i++)
    {
        c[startindex++]=a[index];
    }
}

int main()
{
    int n = 4, m = 2;
    char a[2][4]={{'p','c','a','p'},{'e','x','a','m'}};
    int b[2][4] = {{1, 2, 4, 3}, {2, 4, 3, 2}};
    char *d_a, *d_c;
    int *d_b;
   
    size_t isize = m * n * sizeof(int);
    size_t csize = m * n * sizeof(char);
    int sum=0;
    for(int i=0;i<m;i++)
    for(int j=0;j<n;j++)
    sum+=b[i][j];
    char c[sum];
    hipMalloc((void**)&d_a, csize);
    hipMalloc((void**)&d_b, isize);
    hipMalloc((void**)&d_c, sum*sizeof(char));
    hipMemcpy(d_a, a, csize, hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, isize, hipMemcpyHostToDevice);

    MatModify<<<m, n>>>(d_a, d_b, d_c, n, m);
    hipMemcpy(c, d_c, sum*sizeof(char), hipMemcpyDeviceToHost);
    printf("%s",c);

    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
    return 0;
}
