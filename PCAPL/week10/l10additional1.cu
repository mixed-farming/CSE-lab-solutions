#include <stdio.h>
#include <hip/hip_runtime.h>

__global__ void MatModify(int *a, int *c, int n, int m)
{
    int row = blockIdx.x;
    int col = threadIdx.x;
    int index = row * n + col;
    c[index] = 0;
    for(int i=0;i<n;i++)
    c[index]+=a[row*n+i];
    for(int i=0;i<m;i++)
    c[index]+=a[i*n+col];
}

int main()
{
    int n = 3, m = 2;
    int a[2][3] = {{1, 2, 3}, {4, 5, 6}};
    int c[2][3];
    int *d_a, *d_c;
   
    size_t asize = m * n * sizeof(int);
    hipMalloc((void**)&d_a, asize);
    hipMalloc((void**)&d_c, asize);
    hipMemcpy(d_a, a, asize, hipMemcpyHostToDevice);

    MatModify<<<m, n>>>(d_a, d_c, n, m);
    hipMemcpy(c, d_c, asize, hipMemcpyDeviceToHost);

    printf("Modified matrix:\n");
    for (int i = 0; i < m; i++)
    {
        for (int j = 0; j < n; j++) 
        {
            printf("%d ", c[i][j]);
        }
        printf("\n");
    }

    hipFree(d_a);
    hipFree(d_c);
    return 0;
}
