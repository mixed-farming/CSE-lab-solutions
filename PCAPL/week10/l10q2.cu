    #include <stdio.h>
    #include <hip/hip_runtime.h>
    #include <math.h>

    __global__ void MatModify(int *a,int *c,int n)
    {
        int ele=blockIdx.x*blockDim.x + threadIdx.x;
        c[ele]=1;
        for(int i=0;i<blockIdx.x+1;i++)
        c[ele]*=a[ele];
    }

    int main()
    {
        int n, m;
        printf("Enter dimensions of matrix: ");
        scanf("%d%d", &m, &n);
        int a[m][n],c[m][n];
        printf("Enter the array elements : \n");
        for (int i=0;i<m;i++)
        {
            for (int j=0;j<n;j++) 
            scanf("%d",&a[i][j]);
        }

        int *d_a,*d_c,*a1;
       
        size_t asize;
        asize = m*n*sizeof(int);
        hipMalloc((void**)&d_a,asize);
        hipMalloc((void**)&d_c,asize);
        hipMemcpy(d_a,a,asize,hipMemcpyHostToDevice);
        a1 = (int*)malloc(asize);

        MatModify<<<m,n>>>(d_a,d_c,n);
        hipMemcpy(a1,d_c,asize,hipMemcpyDeviceToHost);
        int k=0;
        printf("Modified matrix:\n");
        for (int i=0;i<m;i++)
        {
            for (int j=0;j<n;j++) 
            {
                c[i][j]=a1[k++];
                printf("%d ",c[i][j]);
            }
            printf("\n");
        }

        hipFree(d_a);hipFree(d_c);
        return 0;
    }

