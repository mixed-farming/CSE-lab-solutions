 #include<stdio.h>
 #include "hip/hip_runtime.h"
 
 __global__ void calc(float* ma, float* a, int mw, float* res){
   int m = blockDim.y, n = blockDim.x, i = threadIdx.y, j = threadIdx.x, sum = 0;
   for(int k = i - mw / 2, p = 0; k <= i + mw / 2; k++, p++)
   for(int l = j - mw / 2, q = 0; l <= j + mw / 2; l++, q++)
   if(k >= 0 && k < m && l < n && l >= 0)
   sum += ma[p * mw + q] * a[k * n + l];
   res[i * n + j] = sum;
 }
 int main(){
   int m, n, mw;
   printf("Enter the array dimensions : ");
   scanf("%d%d", &m, &n);
   float a[m * n], *d_a, res[m * n], *d_res;
   printf("Enter the matrix elements: \n");
   for(int i = 0; i < m * n; i++) scanf("%f", &a[i]);
   printf("Enter the mask width: ");
   scanf("%d", &mw);
   float ma[mw * mw], *d_ma;
   printf("Enter the mask matrix elements: \n");
   for(int i = 0; i < mw * mw; i++) scanf("%f", &ma[i]);
   hipMalloc((void**)&d_a, sizeof(float) * m * n);
   hipMalloc((void**)&d_ma, sizeof(float) * mw * mw);
   hipMalloc((void**)&d_res, sizeof(float) * m * n);
   hipMemcpy(d_a, &a, sizeof(float) * m * n, hipMemcpyHostToDevice);
   hipMemcpy(d_ma, &ma, sizeof(float) * mw * mw, hipMemcpyHostToDevice);
   dim3 bl(n, m);
   calc <<< 1, bl >>> (d_ma, d_a, mw, d_res);
   hipMemcpy(&res, d_res, sizeof(float) * m * n, hipMemcpyDeviceToHost); 
   printf("Resultant matrix is :\n");
   for(int i = 0; i < m; i++){
     for(int j = 0; j < n; j++) printf("%f ", res[i * n + j]);
     printf("\n");
   }
   hipFree(d_a);
   hipFree(d_ma);
   hipFree(d_res);
   return 0;
 }
