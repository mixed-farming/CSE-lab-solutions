#include<stdio.h>
#include<math.h>
#include "hip/hip_runtime.h"

//matrix multiplication using 1d grid and 2d block

__global__ void matMul(int *a,int *b,int *c,int m,int n,int q)
{
    int row = threadIdx.y;
    int col = threadIdx.x;
    int sum = 0;
    int k;

    for (k = 0; k < n; k++) 
    {
        sum += a[row * n + k] * b[k * q + col];
    }
    c[row * q + col] = sum;
}

int main()
{
	int m=3, n=3, p=3, q=3, a[3][3]={{1,2,3},{4,5,6},{7,8,9}},b[3][3]={{0,1,2},{3,4,5},{6,7,8}}, c[3][3];
	printf("a:\n");
	for (int i=0;i<m;i++)
	{
		for (int j=0;j<n;j++) printf(" %d",a[i][j]);
		printf("\n");
	}
	printf("b:\n");
	for (int i=0;i<p;i++)
	{
		for (int j=0;j<q;j++) printf(" %d",b[i][j]);
		printf("\n");
	}

	int *d_a,*d_b,*d_mul; 
    size_t asize,bsize,cmsize;
	asize = m*n*sizeof(int); 
    bsize = p*q*sizeof(int);
    cmsize = m*q*sizeof(int);
	hipMalloc((void**)&d_a,asize); 
    hipMalloc((void**)&d_b,bsize);
    hipMalloc((void**)&d_mul,cmsize);
	hipMemcpy(d_a,a,asize,hipMemcpyHostToDevice);
	hipMemcpy(d_b,b,bsize,hipMemcpyHostToDevice);
    dim3 block(n,m,1);
	matMul<<<1,block>>>(d_a,d_b,d_mul,m,n,q);
	hipMemcpy(c,d_mul,cmsize,hipMemcpyDeviceToHost);
	printf("\na*b:\n");
	for (int i=0;i<m;i++)
	{
		for (int j=0;j<n;j++) 
            printf(" %d",c[i][j]);
		printf("\n");
	}

	hipFree(d_a); hipFree(d_b);hipFree(d_mul);
	return 0;
}
