#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>

__global__ void kernel(char* sin, int* sin_len, char* sout)
{
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    int si = 0; // start index
    for (int i = 0; i < idx; i++)
    {
        si += (*sin_len)-i;
    }
    int total_chars = (*sin_len) - idx;
    for (int i = 0; i < total_chars; i++)
    {
        sout[si++] = sin[i];
    }
}

int main()
{
    char sin[100],sout[100];
    printf("Enter the string : ");
    scanf("%s",sin);

    int sin_len = strlen(sin);
    int sout_len = 0;

    for (int i = 0; i < sin_len; i++)
    {
        sout_len += (i+1);
    }

    char* d_sin;
    int* d_sin_len;
    char* d_sout;

    hipMalloc((void**) &d_sin, sin_len * sizeof(char));
    hipMalloc((void**) &d_sin_len, sizeof(int));
    hipMalloc((void**) &d_sout, (sout_len + 1) * sizeof(char));

    hipMemcpy(d_sin, sin, sin_len * sizeof(char), hipMemcpyHostToDevice);
    hipMemcpy(d_sin_len, &sin_len, sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_sout, sout, (sout_len + 1) * sizeof(char), hipMemcpyHostToDevice);

    kernel<<<1, sin_len>>>(d_sin, d_sin_len, d_sout);
    hipMemcpy(sout, d_sout, (sout_len + 1) * sizeof(char), hipMemcpyDeviceToHost);
    sout[sout_len] = '\0';

    printf("Sin: %s\n", sin);
    printf("Sout: %s\n", sout);

    hipFree(d_sin);
    hipFree(d_sin_len);
    hipFree(d_sout);

    return 0;
}