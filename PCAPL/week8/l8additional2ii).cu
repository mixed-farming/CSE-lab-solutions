#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>

//number of threads = length of the string

__global__ void kernel(char* sin, int n, char* sout)
{
    int si = threadIdx.x,len=blockDim.x; 
    for (int i = 0; i < n; i++)
    {
        sout[si+i*len] = sin[si];
    }
}

int main()
{
    char *s="hello";
    int n=3;
    char *out = (char*) malloc((strlen(s) + 1) * sizeof(char) * n);

    int len = strlen(s);
    int out_len=len*n;

    char* d_s;
    char* d_out; 

    hipMalloc((void**) &d_s, len * sizeof(char));
    hipMalloc((void**) &d_out, (out_len + 1) * sizeof(char));
    hipMemcpy(d_s, s, len * sizeof(char), hipMemcpyHostToDevice);

    kernel<<<1, strlen(s)>>>(d_s, n, d_out);
    hipMemcpy(out, d_out, (out_len + 1) * sizeof(char), hipMemcpyDeviceToHost);
    out[out_len] = '\0';

    printf("Sin: %s\n", s);
    printf("Sout: %s\n", out); 

    hipFree(d_s);
    hipFree(d_out);
    free(out);

    return 0;
}