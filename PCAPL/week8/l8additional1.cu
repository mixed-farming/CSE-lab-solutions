#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <string.h>

__global__ void function(char* a, int* b, int len)
{
    int idx = threadIdx.x;
    int start = b[idx] + 1;
    int end = b[idx + 1];
    
    for (int i = start, j = end-1; i < j; i++, j--) {
        char temp = a[i];
        a[i] = a[j];
        a[j] = temp;
    }
}

int main()
{
    char *a = "i erocs lluf ni lpacp mesdne";
    char *c = (char*) malloc((strlen(a) + 1) * sizeof(char));
    int len = strlen(a);
    size_t s = len * sizeof(char);

    int b[10];
    int k = 0;
    b[k++] = -1;
    
    for (int i = 0; i < len; i++) {
        if (a[i] == ' ') {
            b[k++] = i;
        }
    }
    b[k++] = len;
    
    char *d_a;
    int *d_b;
    int words=k+1;//words=number of spaces+1
    
    hipMalloc((void**)&d_a, s);
    hipMalloc((void**)&d_b, words * sizeof(int));
    
    hipMemcpy(d_a, a, s, hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, words * sizeof(int), hipMemcpyHostToDevice);
    
    function<<<1, words>>>(d_a, d_b, len);
    
    hipMemcpy(c, d_a, s, hipMemcpyDeviceToHost);
    printf("%s\n", c);
    
    hipFree(d_a);
    hipFree(d_b);
    free(c);
 
    return 0;
}
