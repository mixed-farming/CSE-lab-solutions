#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>

__global__ void kernel(char* sin, char* sout)
{
    int id= threadIdx.x;
    int si = (id*(id+1))/2;
    for (int i = 0; i < id+1; i++)
    {
        sout[si++] = sin[id];
    }
}

int main()
{
    char *s="hai";
    int len = strlen(s);
    int out_len=(len*(len+1))/2;
    char *out = (char*) malloc(sizeof(char) * out_len);

    char* d_s;
    char* d_out; 

    hipMalloc((void**) &d_s, len * sizeof(char));
    hipMalloc((void**) &d_out, (out_len + 1) * sizeof(char));
    hipMemcpy(d_s, s, len * sizeof(char), hipMemcpyHostToDevice);

    kernel<<<1, strlen(s)>>>(d_s, d_out);
    hipMemcpy(out, d_out, (out_len + 1) * sizeof(char), hipMemcpyDeviceToHost);
    out[out_len] = '\0';

    printf("Sin: %s\n", s);
    printf("Sout: %s\n", out); 

    hipFree(d_s);
    hipFree(d_out);
    free(out);

    return 0;
}