#include "hip/hip_runtime.h"

#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#define N 1024

__global__ void CUDACount(char* A, unsigned int *d_count){
    int i = threadIdx.x;
    if(A[i]=='a')
    atomicAdd(d_count,1);
} 

int main() {
    char A[N];
    char *d_A;
    unsigned int *count=0,*d_count,result;
    printf("Enter a string : ");
    scanf("%s",A);
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);
    hipMalloc((void**)&d_A, strlen(A)*sizeof(char));
    hipMalloc((void **)&d_count,sizeof(unsigned int));
    hipMemcpy(d_A, A, strlen(A)*sizeof(char), hipMemcpyHostToDevice);
    hipMemcpy(d_count,count,sizeof(unsigned int),hipMemcpyHostToDevice);

    CUDACount<<<1,strlen(A)>>>(d_A,d_count);

    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    float elapsedTime;
    hipEventElapsedTime(&elapsedTime, start, stop);
    hipMemcpy(&result, d_count, sizeof(unsigned int), hipMemcpyDeviceToHost);
    printf("Total occurences of a=%u\n",result);
    printf("Time Taken=%f",elapsedTime);
    hipFree(d_A);
    hipFree(d_count);
    printf("\n");
    return 0;
}