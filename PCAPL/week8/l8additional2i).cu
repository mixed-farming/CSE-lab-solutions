#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>

//number of threads = number of repetitions of the word

__global__ void kernel(char* sin, int sin_len, char* sout)
{
    int si = threadIdx.x*sin_len; 
    for (int i = 0; i < sin_len; i++)
    {
        sout[si++] = sin[i];
    }
}

int main()
{
    char *s="hello";
    int n=3;
    char *out = (char*) malloc((strlen(s) + 1) * sizeof(char) * n);

    int len = strlen(s);
    int out_len=len*n;

    char* d_s;
    char* d_out; 

    hipMalloc((void**) &d_s, len * sizeof(char));
    hipMalloc((void**) &d_out, (out_len + 1) * sizeof(char));
    hipMemcpy(d_s, s, len * sizeof(char), hipMemcpyHostToDevice);

    kernel<<<1, n>>>(d_s, strlen(s), d_out);
    hipMemcpy(out, d_out, (out_len + 1) * sizeof(char), hipMemcpyDeviceToHost);
    out[out_len] = '\0';

    printf("Sin: %s\n", s);
    printf("Sout: %s\n", out); 

    hipFree(d_s);
    hipFree(d_out);
    free(out);

    return 0;
}