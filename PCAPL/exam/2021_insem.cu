//Given an NXN matrix mat, where N is an even number and N >= 4, 
//write a CUDA program to find major diagonal sums of partial matrices in the four different quadrants 
//(vertical and horizontal lines are drawn to partition the matrix into 4 quadrants) of mat in parallel as shown below in sample Input/Output. 
//Use 2X2 grid with 1 thread per block. The first parameter to the kernel is the matrix and the second is an 1D array of size 4 to store 4 different diagonal sums 
//computed by 4 different threads of 4 different blocks. Pass any other parameters if necessary. Read N and mat contents in the host code.  
//Display the matrix mat and the diagonal sums in the host code. The code should work for any value of N satisfying the condition mentioned above. 
//Use dynamic allocation for all the host arrays.
/*
i/p:
0 1 2  3 4 5            
1 2 3  4 5 6            
2 3 4  5 6 7            
3 4 5  6 7 8            
4 5 6  7 8 9            
5 6 7  8 9 10
o/p:
Diagonal sums from threads of different blocks:
From block 0 diagonal Sum = 6  
From block 1 diagonal Sum = 15
From block 2 diagonal Sum = 15 
From block 3 diagonal Sum = 24
*/

#include <stdio.h>
#include <hip/hip_runtime.h>

__global__ void diagonalSumKernel(int* mat, int* diagSums, int n) {
    int bid_x = blockIdx.x;
    int bid_y = blockIdx.y;
    int rowStart = bid_x * n/2;
    int colStart = bid_y * n/2;
    int sum = 0;
    for (int i = 0; i < n/2 ; i++) {
        sum += mat[(rowStart + i) * n + colStart + i];
    }
    diagSums[bid_x * 2 + bid_y] = sum;
}

int main() {
    int n=6, *diagSums;
    int size = n * n * sizeof(int);
    int size_ds = 4 * sizeof(int);

    // Allocate memory for matrix and diagonal sums array on host
    diagSums = (int*)malloc(size_ds);
    int mat[6][6]={{0,1,2,3,4,5},{1,2,3,4,5,6},{2,3,4,5,6,7},{3,4,5,6,7,8},{4,5,6,7,8,9},{5,6,7,8,9,10}};

    int *d_mat, *d_diagSums;
    hipMalloc((void**)&d_mat, size);
    hipMalloc((void**)&d_diagSums, size_ds);
    hipMemcpy(d_mat, mat, size, hipMemcpyHostToDevice);
    dim3 grid(2, 2, 1);
    diagonalSumKernel<<<grid, 1>>>(d_mat, d_diagSums, n);

    hipMemcpy(diagSums, d_diagSums, size_ds, hipMemcpyDeviceToHost);

    printf("\nDiagonal sums from threads of different blocks:\n");
    for (int i = 0; i < 4; i++) {
        printf("From block %d diagonal Sum = %d\n", i, diagSums[i]);
    }

    free(diagSums);
    hipFree(d_mat);
    hipFree(d_diagSums);

    return 0;
}
