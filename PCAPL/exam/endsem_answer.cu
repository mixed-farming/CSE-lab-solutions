#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <hip/hip_runtime.h>

__global__ void modify(char* s,int* pos, char m, int n)
{
  int idx=blockIdx.x*blockDim.x*blockDim.y+threadIdx.y*blockDim.x+threadIdx.x;
  int start=pos[idx]+1;
  int end=pos[idx+1];
  if(idx<pos[idx+1]-pos[idx])
  {
    if(m=='R')
    {
      for(int i=0;i<n;i++)
      {
        char temp=s[end-1];
        for(int j=end-2;j>=start;j--)
        {
          s[j+1]=s[j];
        }
        s[start]=temp;
      }
    }
    else if(m=='L')
    {
      for(int i=0;i<n;i++)
      {
        char temp=s[start];
        for(int j=start;j<end;j++)
        {
          s[j]=s[j+1];
        }
        s[end-1]=temp;
      }
    }
  }
}

int main()
{
  char *s, c;
  int n;
  s=(char*)malloc(sizeof(char)*100);
  printf("Enter the string : ");
  scanf("%[^\n]%*c",s); //keep scanning every character(*c -> zero or more) until it's not '\n'
  
  printf("Enter the rotation orientation : ");
  scanf("%c",&c);
  
  printf("Enter the number of rotations : ");
  scanf("%d",&n);
  
  int l=strlen(s);
  int b[100],k=0;
  b[k++]=-1;
  for(int i=0;i<l;i++)
  {
    if(s[i]==' ')
      b[k++]=i;
  }
  b[k++]=l;
  printf("Total number of words = %d",k-1);
  printf("\nLength of each word = ");
  int maxi=-1;
  for(int i=0;i<k-1;i++)
  {
    int d=b[i+1]-b[i];
    if(d>maxi)
      maxi=d;
    printf("%d ",d-1);
  }
  
  printf("\nLength of the longest word = %d\n",maxi-1);
  
  char *d_s;
  int *d_pos;
  hipMalloc((void**)&d_s,sizeof(char)*l);
  hipMalloc((void**)&d_pos,sizeof(int)*k);
  hipMemcpy(d_s,s,sizeof(char)*l,hipMemcpyHostToDevice);
  hipMemcpy(d_pos,b,sizeof(int)*k,hipMemcpyHostToDevice);
  dim3 block(2,maxi/2+1,1);
  dim3 grid(k,1,1);
  
  modify<<<grid,block>>>(d_s,d_pos,c,n);
  char* result;
  result=(char*)malloc(sizeof(char)*l);
  hipMemcpy(result,d_s,sizeof(char)*l,hipMemcpyDeviceToHost);
  result[l]='\0';
  printf("Output string : %s\n",result);
 
  hipFree(d_pos);
  hipFree(d_s);
  free(result);
  
  return 0;
}      
