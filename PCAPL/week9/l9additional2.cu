#include "hip/hip_runtime.h"

#include <stdio.h>
#include <stdlib.h>
__global__ void modify(int *a, int *t, int n)
{
  int row=blockIdx.x, col=threadIdx.x;
  int index=row*n+col;
  if(row==col)
  t[index]=0;
 else if(row<col)
 {
     t[index]=1;
      for(int i=a[index];i>1;i--)
      {
          t[index]*=i;
      }
 }
 else
 {
     t[index]=0;
      while(a[index])
      {
          t[index]+=(a[index]%10);
          a[index]/=10;
      }
 }
}

int main(void)
{
  int a[3][3]={{4,5,6},{49,50,7},{79,84,91}},t[3][3], n=3,i,j;
  int *d_a,*d_t;

  size_t size1=n*n*sizeof(int);
  hipMalloc((void**)&d_a,size1);
  hipMalloc((void**)&d_t,size1);
  hipMemcpy(d_a,a,size1,hipMemcpyHostToDevice);
  dim3 grid(n,1,1);
  dim3 block(n,1,1);
  modify<<<grid, block>>>(d_a, d_t, n);
  hipMemcpy(t,d_t,size1,hipMemcpyDeviceToHost);
  printf("Modified matrix is:\n");
  for(i=0;i<n;i++)
  {
  for(j=0;j<n;j++)
  printf("%d\t",t[i][j]);
  printf("\n");
  }
  hipFree(d_a);
  hipFree(d_t);
  return 0;
}
