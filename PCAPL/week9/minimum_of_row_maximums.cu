#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

//minimum of row maximums

__global__ void row_max(int *a,int *rowmax)
{
    int idx=threadIdx.x+blockIdx.x*blockDim.x;
    atomicMax(&rowmax[blockIdx.x],a[idx]);
}

__global__ void minmax(int *rowmax,int *value)
{
    int idx=threadIdx.x;
    atomicMin(value,rowmax[idx]);
}

int main()
{
 int n=4;
 int a[4][4]={{5,6,7,4},{2,5,8,2},{9,7,4,1},{2,5,5,2}}, *d_a, *d_rowmax, rowmax[4];
 hipMalloc((void**)&d_a, sizeof(int)*n*n);
 hipMalloc((void**)&d_rowmax, sizeof(int)*n);
 hipMemcpy(d_a,a,sizeof(int)*n*n,hipMemcpyHostToDevice);
 
 row_max<<<n,n>>>(d_a,d_rowmax);

 hipMemcpy(rowmax, d_rowmax, sizeof(int)*n, hipMemcpyDeviceToHost);

  printf("Row maximums : ");
 for(int i=0;i<n;i++)
 {
     printf("%d ",rowmax[i]);
 }
  
  int x=1000,*d_value;
  hipMalloc((void**)&d_value, sizeof(int));
  hipMemcpy(d_value,&x,sizeof(int),hipMemcpyHostToDevice);
  minmax<<<1,n>>>(d_rowmax,d_value);
  hipMemcpy(&x,d_value,sizeof(int),hipMemcpyDeviceToHost);
  printf("\nMinimum of row maximums : %d",x);

  hipFree(d_a); hipFree(d_rowmax);
  return 0;
}