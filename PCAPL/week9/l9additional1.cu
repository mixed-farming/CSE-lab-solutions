#include "hip/hip_runtime.h"

#include <stdio.h>
#include <stdlib.h>
__global__ void modify(int *a, int *t, int n,int m)
{
    int row=blockIdx.x, col=threadIdx.x;
    int index=row*n+col;
 int sum=0;
    if(a[index]%2==0)
    {
        for(int i=0;i<n;i++)
        {
            sum+=a[row*n+i];
        }
     t[index]=sum;
    }
    else
 {
   for(int i=0;i<m;i++)
   {
       sum+=a[i*n+col];
   }
  t[index]=sum;
 }

}

int main(void)
{
  int a[2][3]={{1,2,3},{4,5,6}},t[2][3], m=2,n=3,i,j;
  int *d_a,*d_t;

  size_t size1=m*n*sizeof(int);
  hipMalloc((void**)&d_a,size1);
  hipMalloc((void**)&d_t,size1);
  hipMemcpy(d_a,a,size1,hipMemcpyHostToDevice);
  modify<<<m,n>>>(d_a,d_t,n,m);
  hipMemcpy(t,d_t,size1,hipMemcpyDeviceToHost);
  printf("Result vector is:\n");
  for(i=0;i<m;i++)
  {
  for(j=0;j<n;j++)
  printf("%d\t",t[i][j]);
  printf("\n");
  }
  hipFree(d_a);
  hipFree(d_t);
  return 0;
}
