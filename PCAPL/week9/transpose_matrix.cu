#include "hip/hip_runtime.h"

#include <stdio.h>
#include <stdlib.h>
__global__ void transpose(int *a, int *t)
{
int n=threadIdx.x, m=blockIdx.x, size=blockDim.x, size1=gridDim.x;
t[n*size1+m]=a[m*size+n];
}

int main(void)
{
  int a[2][3]={{1,2,3},{4,5,6}},t[3][2], m=2,n=3,i,j;
  int *d_a,*d_t;

  size_t size1=m*n*sizeof(int);
  hipMalloc((void**)&d_a,size1);
  hipMalloc((void**)&d_t,size1);
  hipMemcpy(d_a,a,size1,hipMemcpyHostToDevice);
  transpose<<<m,n>>>(d_a,d_t);
  hipMemcpy(t,d_t,size1,hipMemcpyDeviceToHost);
  printf("Result vector is:\n");
  for(i=0;i<n;i++)
  {
  for(j=0;j<m;j++)
  printf("%d\t",t[i][j]);
  printf("\n");
  }
  hipFree(d_a);
  hipFree(d_t);
  return 0;
}
