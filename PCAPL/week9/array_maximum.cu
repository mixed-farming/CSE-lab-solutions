#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

//array maximum

__global__ void find_max(int *arr, int *max_val)
{
    int tid = threadIdx.x;
    atomicMax(max_val, arr[tid]);
}

int main()
{
    int n=10;
    int arr[10]={10,30,90,40,50,20,70,80,60,37}, *d_arr, *d_max;
    int max_val = 0;

    hipMalloc(&d_arr, n * sizeof(int));
    hipMalloc(&d_max, sizeof(int));

    hipMemcpy(d_arr, arr, n * sizeof(int), hipMemcpyHostToDevice);

    find_max<<<1, 256>>>(d_arr, d_max);

    hipMemcpy(&max_val, d_max, sizeof(int), hipMemcpyDeviceToHost);

    for(int i=0;i<n;i++)
    {
        printf("%d ",arr[i]);
    }
    printf("\n\nMax value: %d\n", max_val);

    hipFree(d_arr);
    hipFree(d_max);

    return 0;
}
