#include "hip/hip_runtime.h"
%%cu
#include<stdio.h>
#include<math.h>
#include "hip/hip_runtime.h"
__global__ void matRowAdd(int *a,int *b,int *c,int m,int n)
{
	int row=threadIdx.x;
	for (int i=0;i<n;i++) c[row*n+i]=a[row*n+i]+b[row*n+i];
}
__global__ void matColAdd(int *a,int *b,int *c,int m,int n)
{
	int col=threadIdx.x;
	for (int i=0;i<m;i++) c[i*n+col]=a[i*n+col]+b[i*n+col];
}
__global__ void matAdd(int *a,int *b,int *c,int m,int n)
{
	int row=blockIdx.y*blockDim.y + threadIdx.y;
	int col=blockIdx.x*blockDim.x + threadIdx.x;
	if (row<m && col<n) c[row*n+col]=a[row*n+col]+b[row*n+col];
}
__global__ void matRowMul(int *a,int *b,int *c,int n,int q)
{
	int rowidA=threadIdx.x,sum,k;
	for (int colidB=0;colidB<q;colidB++)
	{
		sum=0;
		for (k=0;k<n;k++) sum=sum+a[rowidA*n+k]*b[k*q+colidB];
		c[rowidA*q+colidB]=sum;
	}
}
__global__ void matColMul(int *a,int *b,int *c,int m,int n)
{
	int colidB=threadIdx.x,q=blockDim.x,sum,k;
	for (int rowidA=0;rowidA<m;rowidA++)
	{
		sum=0;
		for (k=0;k<n;k++) sum=sum+a[rowidA*n+k]*b[k*q+colidB];
		c[rowidA*q+colidB]=sum;
	}
}
__global__ void matMul(int *a,int *b,int *c,int m,int n,int q)
{
	int row=threadIdx.y;
	int col=threadIdx.x;
	int sum=0,k;
	if (row<m && col<q) for (k=0;k<n;k++) sum=sum+a[row*n+k]*b[k*q+col];
	c[row*q+col]=sum;
}
int main()
{
	int m=3, n=3, p=3, q=3, blocksize=2, a[3][3]={{1,2,3},{4,5,6},{7,8,9}},
	b[3][3]={{0,1,2},{3,4,5},{6,7,8}}, c[3][3];
	printf("a:\n");
	for (int i=0;i<m;i++)
	{
		for (int j=0;j<n;j++) printf(" %d",a[i][j]);
		printf("\n");
	}
	printf("b:\n");
	for (int i=0;i<p;i++)
	{
		for (int j=0;j<q;j++) printf(" %d",b[i][j]);
		printf("\n");
	}

	int *d_a,*d_b,*d_add,*d_mul; size_t asize,bsize,casize,cmsize;
	asize = m*n*sizeof(int); bsize = p*q*sizeof(int);
	casize = m*n*sizeof(int); cmsize = m*q*sizeof(int);
	hipMalloc((void**)&d_a,asize); hipMalloc((void**)&d_b,bsize);
	hipMalloc((void**)&d_add,casize); hipMalloc((void**)&d_mul,cmsize);
	hipMemcpy(d_a,a,asize,hipMemcpyHostToDevice);
	hipMemcpy(d_b,b,bsize,hipMemcpyHostToDevice);
	dim3 block(2,2,1);

	matRowAdd<<<1,m>>>(d_a,d_b,d_add,m,n);
	hipMemcpy(c,d_add,casize,hipMemcpyDeviceToHost);
	printf("1.(a) a+b:\n");
	for (int i=0;i<m;i++)
	{
		for (int j=0;j<n;j++) printf(" %d",c[i][j]);
		printf("\n");
	}

	matColAdd<<<1,n>>>(d_a,d_b,d_add,m,n);
	hipMemcpy(c,d_add,casize,hipMemcpyDeviceToHost);
	printf("1.(b) a+b:\n");
	for (int i=0;i<m;i++)
	{
		for (int j=0;j<n;j++) printf(" %d",c[i][j]);
		printf("\n");
	}

	dim3 grid(ceil(m/(float)blocksize),ceil(n/(float)blocksize),1);
	matAdd<<<grid,block>>>(d_a,d_b,d_add,m,n);
	hipMemcpy(c,d_add,casize,hipMemcpyDeviceToHost);
	printf("1.(c) a+b:\n");
	for (int i=0;i<m;i++)
	{
		for (int j=0;j<n;j++) printf(" %d",c[i][j]);
		printf("\n");
	}

	matRowMul<<<1,m>>>(d_a,d_b,d_mul,n,q);
	hipMemcpy(c,d_mul,cmsize,hipMemcpyDeviceToHost);
	printf("2.(a) a*b:\n");
	for (int i=0;i<m;i++)
	{
		for (int j=0;j<n;j++) printf(" %d",c[i][j]);
		printf("\n");
	}

	matColMul<<<1,q>>>(d_a,d_b,d_mul,m,n);
	hipMemcpy(c,d_mul,cmsize,hipMemcpyDeviceToHost);
	printf("2.(b) a*b:\n");
	for (int i=0;i<m;i++)
	{
		for (int j=0;j<n;j++) printf(" %d",c[i][j]);
		printf("\n");
	}

	dim3 mulgrid(ceil((q+blocksize-1)/blocksize),ceil((m+blocksize-1)/blocksize),1);
	matMul<<<mulgrid,block>>>(d_a,d_b,d_mul,m,n,q);
	hipMemcpy(c,d_mul,cmsize,hipMemcpyDeviceToHost);
	printf("2.(c) a*b:\n");
	for (int i=0;i<m;i++)
	{
		for (int j=0;j<n;j++) printf(" %d",c[i][j]);
		printf("\n");
	}

	hipFree(d_a); hipFree(d_b); hipFree(d_add); hipFree(d_mul);
	return 0;
}